#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#define DEBUG_WRITING (0)

#define WID_SRC (768)
#define HEI_SRC (576)
#define CHN_SRC (3)
#define WID_SIZED (608)
#define HEI_SIZED (608)
#define WID_DST (19)
#define HEI_DST (19)
#define CHN_DST (425)
#define MAX_OUT (11829248)

#define SIZE_MAX_WORKSPACE (30000000)
#define NUM_LAYER (32)

#define BLOCK (512)

#define CHK_INTER_LAYER (0)
#define ACCEPTABLE_DIFF (0.005f)

#if (1 == DEBUG_WRITING)
FILE *fp_fprintf_debug;
#endif

typedef struct {
    int w;
    int h;
    int c;
    float *data;
} image;

typedef enum{
    LOGISTIC, RELU, RELIE, LINEAR, RAMP, TANH, PLSE, LEAKY, ELU, LOGGY, STAIR, HARDTAN, LHTAN
} ACTIVATION;

static int sa_typ_s32[NUM_LAYER];
static int sa_wid_s32[NUM_LAYER];
static int sa_hei_s32[NUM_LAYER];
static int sa_chn_s32[NUM_LAYER];
static int sa_ker_s32[NUM_LAYER];
static int sa_pad_s32[NUM_LAYER];
static int sa_ibn_s32[NUM_LAYER];
static int sa_nwe_s32[NUM_LAYER];
static unsigned char sa_image_in_0_u08[WID_SRC * HEI_SRC * CHN_SRC];
static unsigned char sa_image_in_1_u08[WID_SRC * HEI_SRC * CHN_SRC];
#if (1 == CHK_INTER_LAYER)
static float sa_image_sized_f32[WID_SIZED * HEI_SIZED * CHN_SRC];
#endif
static float sa_tmp_buf_f32[SIZE_MAX_WORKSPACE];
static float sa_out_f32[WID_DST * HEI_DST * CHN_DST];
static float *spa_out_f32[NUM_LAYER];
static float *spa_weights_f32[NUM_LAYER];
static float *spa_mean_f32[NUM_LAYER];
static float *spa_variance_f32[NUM_LAYER];
static float *spa_scales_f32[NUM_LAYER];
static float *spa_biases_f32[NUM_LAYER];
static float *sp_gpu_int_0_f32;
static float *sp_gpu_int_1_f32;
static float *sp_gpu_int_16_f32;
static float *sp_gpu_int_24_f32;
static float *sp_gpu_int_27_f32;
static float *sp_gpu_weights_f32[NUM_LAYER];
static float *sp_gpu_mean_f32[NUM_LAYER];
static float *sp_gpu_variance_f32[NUM_LAYER];
static float *sp_gpu_scales_f32[NUM_LAYER];
static float *sp_gpu_biases_f32[NUM_LAYER];
static float *sp_gpu_input_f32;
static float *sp_gpu_workspace_f32;
static float sa_ref_sized_f32[WID_SIZED * HEI_SIZED * CHN_SRC];
static float *spa_ref_f32[NUM_LAYER];

static void yolo_main(float *p_out_f32, unsigned char *p_image_in_u08);
static float *cuda_make_array(float *x, size_t n);
static void check_error(hipError_t status);
static void fill_gpu(int N, float ALPHA, float * X, int INCX);
static dim3 cuda_gridsize(size_t n);
static void im2col_gpu(float *im, int channels, int height, int width, int ksize, int stride, int pad, float *data_col);
static void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, float *A_gpu, int lda, float *B_gpu, int ldb, float BETA, float *C_gpu, int ldc);
static void forward_convolutional_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_weights_gpu, float *workspace_gpu, float *mean_gpu, float *variance_gpu, float *scales_gpu, float *biases_gpu, int l_outputs, int l_n, int l_size, int l_c, int l_out_w, int l_out_h, int l_w, int l_h, int l_stride, int l_pad, int l_batch_normalize, ACTIVATION l_activation);
static void convolution_ref_c(float * __restrict p_out_f32, const float * __restrict p_in_f32, const float * __restrict p_weights_f32, const int chn_in_s32, const int wid_in_s32, const int hei_in_s32, const int chn_out_s32, const int wid_out_s32, const int hei_out_s32, const int ker_s32, const int pad_s32);
static hipblasHandle_t blas_handle();
static int cuda_get_device();
static void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial);
static void scale_bias_gpu(float *output, float *biases, int batch, int n, int size);
static void add_bias_gpu(float *output, float *biases, int batch, int n, int size);
static void activate_array_gpu(float *x, int n, ACTIVATION a);
static void forward_maxpool_layer_gpu(float *l_output_gpu, float *input_gpu, int layer_out_w, int layer_out_h, int layer_batch, int layer_w, int layer_h, int layer_c, int layer_stride, int layer_size, int layer_pad);
static void forward_route_layer_25_gpu(float *l_output_gpu, float *input_l16);
static void forward_route_layer_28_gpu(float *l_output_gpu, float *input_l27, float *input_l24);
static void copy_gpu(int N, float * X, int INCX, float * Y, int INCY);
static void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY);
static void forward_reorg_layer_gpu(float *l_output_gpu, float *input_gpu, int l_w, int l_h, int l_c, int l_batch, int l_stride);
static void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out);
static void forward_region_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_output, int l_batch, int l_inputs, int l_n, int l_w, int l_h, int l_coords, int l_background, int l_classes, int l_outputs);
static int entry_index(int l_w, int l_h, int l_outputs, int l_coords, int l_classes, int batch, int location, int entry);
static void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output);
static void cuda_pull_array(float *x_gpu, float *x, size_t n);
#if (1 == CHK_INTER_LAYER)
static void check_intermediate_layer_results(int l);
#endif

static unsigned char *sp_gpu_image_in_u08;
static float *sp_gpu_image_in_f32;
static float *sp_gpu_resized_f32;
static float *sp_gpu_part_f32;

int main(void)
{
    FILE *fp;
    FILE *fp_results;
    FILE *fp_weights;
    FILE *fp_mean;
    FILE *fp_variance;
    FILE *fp_scales;
    FILE *fp_biases;
    FILE *fp_netinfo;
    int i, j, k;
    size_t fread_return;
    clock_t clk_srt, clk_end;
    hipError_t status;
    int nDevices_s32;

    hipGetDeviceCount(&nDevices_s32);

    for (i = 0; i < nDevices_s32; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf(" Device name: %s\n", prop.name);
        printf(" Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf(" Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf(" Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf(" totalGlobalMem: %ld\n", prop.totalGlobalMem);
        printf(" sharedMemPerBlock: %ld\n", prop.sharedMemPerBlock);
        printf(" regsPerBlock: %d\n", prop.regsPerBlock);
        printf(" warpSize: %d\n", prop.warpSize);
    }

    printf("\nyolo reference CUDA code by Hyuk Lee\n");

#if (1 == DEBUG_WRITING)
    fp_fprintf_debug = fopen("ref_c_debug.txt", "w");
#endif

    /* read net info */
    fp_netinfo = fopen("yolo_gpu_netinfo.bin", "rb");
    if(NULL == fp_netinfo)
    {
        printf("yolo_gpu_netinfo fopen error\n");
        return -1;
    }
    /* load weights */
    fp_weights = fopen("yolo_gpu_weights.bin", "rb");
    if(NULL == fp_weights)
    {
        printf("yolo_gpu_weights fopen error\n");
        return -1;
    }
    fp_mean = fopen("yolo_gpu_mean.bin", "rb");
    if(NULL == fp_mean)
    {
        printf("yolo_gpu_mean fopen error\n");
        return -1;
    }
    fp_variance = fopen("yolo_gpu_variance.bin", "rb");
    if(NULL == fp_variance)
    {
        printf("yolo_gpu_variance fopen error\n");
        return -1;
    }
    fp_scales = fopen("yolo_gpu_scales.bin", "rb");
    if(NULL == fp_scales)
    {
        printf("yolo_gpu_scales fopen error\n");
        return -1;
    }
    fp_biases = fopen("yolo_gpu_biases.bin", "rb");
    if(NULL == fp_biases)
    {
        printf("yolo_gpu_biases fopen error\n");
        return -1;
    }
    /* load ref data */
    fp = fopen("yolo_image_sized.bin", "rb");
    if(NULL == fp)
    {
        printf("yolo_image_sized fopen error\n");
        return -1;
    }
    fread_return = fread(sa_ref_sized_f32, WID_SIZED * HEI_SIZED * CHN_SRC, sizeof(float), fp);
    fclose(fp);
    fp_results = fopen("yolo_gpu_intermediate_results.bin", "rb");
    if(NULL == fp_results)
    {
        printf("yolo_gpu_results fopen error\n");
        return -1;
    }
    status = hipMalloc((void **)&sp_gpu_int_0_f32, MAX_OUT * sizeof(float));
    status = hipMalloc((void **)&sp_gpu_int_1_f32, MAX_OUT * sizeof(float));
    status = hipMalloc((void **)&sp_gpu_int_16_f32, MAX_OUT * sizeof(float));
    status = hipMalloc((void **)&sp_gpu_int_24_f32, MAX_OUT * sizeof(float));
    status = hipMalloc((void **)&sp_gpu_int_27_f32, MAX_OUT * sizeof(float));
    for(i = 0; i < NUM_LAYER; i++)
    {
        fread_return = fread(&sa_typ_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_wid_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_hei_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_chn_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_ker_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_pad_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_ibn_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_nwe_s32[i], 1, sizeof(int), fp_netinfo);        
        spa_out_f32[i] = (float *)malloc(sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i] * sizeof(float));
        if(sa_typ_s32[i] == 0) /* convolutional */
        {
            spa_weights_f32[i] = (float *)malloc(sa_nwe_s32[i] * sizeof(float));
            fread_return = fread(spa_weights_f32[i], sa_nwe_s32[i], sizeof(float), fp_weights);
            sp_gpu_weights_f32[i] = cuda_make_array(spa_weights_f32[i], sa_nwe_s32[i]);
            if(sa_ibn_s32[i] == 1)
            {
                spa_mean_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                spa_variance_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                spa_scales_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                fread_return = fread(spa_mean_f32[i], sa_chn_s32[i], sizeof(float), fp_mean);
                fread_return = fread(spa_variance_f32[i], sa_chn_s32[i], sizeof(float), fp_variance);
                fread_return = fread(spa_scales_f32[i], sa_chn_s32[i], sizeof(float), fp_scales);
                sp_gpu_mean_f32[i] = cuda_make_array(spa_mean_f32[i], sa_chn_s32[i]);
                sp_gpu_variance_f32[i] = cuda_make_array(spa_variance_f32[i], sa_chn_s32[i]);
                sp_gpu_scales_f32[i] = cuda_make_array(spa_scales_f32[i], sa_chn_s32[i]);
            }
            spa_biases_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
            fread_return = fread(spa_biases_f32[i], sa_chn_s32[i], sizeof(float), fp_biases);
            sp_gpu_biases_f32[i] = cuda_make_array(spa_biases_f32[i], sa_chn_s32[i]);
        }
        spa_ref_f32[i] = (float *)malloc(sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i] * sizeof(float));
        fread_return = fread(spa_ref_f32[i], sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i], sizeof(float), fp_results);
    }
    fclose(fp_results);
    fclose(fp_weights);
    fclose(fp_mean);
    fclose(fp_variance);
    fclose(fp_scales);
    fclose(fp_biases);
    fclose(fp_netinfo);

    /* read input data */
    fp = fopen("yolo_image_in.bin", "rb");
    if(NULL == fp)
    {
        printf("yolo_image_in fopen error\n");
        return -1;
    }
    fread_return = fread(sa_image_in_0_u08, WID_SRC * HEI_SRC * CHN_SRC, sizeof(unsigned char), fp);
    fclose(fp);

    status = hipMalloc((void **)&sp_gpu_workspace_f32, SIZE_MAX_WORKSPACE * sizeof(float));
    check_error(status);
    sp_gpu_input_f32 = cuda_make_array(sa_tmp_buf_f32, WID_SIZED * HEI_SIZED * CHN_SRC);
    status = hipMalloc((void **)&sp_gpu_image_in_u08, WID_SRC * HEI_SRC * CHN_SRC * sizeof(unsigned char));
    check_error(status);
    status = hipMalloc((void **)&sp_gpu_image_in_f32, WID_SRC * HEI_SRC * CHN_SRC * sizeof(float));
    check_error(status);
    status = hipMalloc((void **)&sp_gpu_resized_f32, 608 * 456 * 3 * sizeof(float));
    check_error(status);
    status = hipMalloc((void **)&sp_gpu_part_f32, 608 * 576 * 3 * sizeof(float));
    check_error(status);

    clk_srt = clock();
    yolo_main(sa_out_f32, sa_image_in_0_u08);
    clk_end = clock();
    printf("yolo 1: %f s\n", (double)(clk_end - clk_srt) / CLOCKS_PER_SEC);

#if 0
    for(k = 0; k < CHN_DST; k++)
    {
        for(j = 0; j < HEI_DST; j++)
        {
            for(i = 0; i < WID_DST; i++)
            {
                if(fabsf(sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST] - spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]) > ACCEPTABLE_DIFF)
                {
                    printf("final results mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST], spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]);
                }
            }
        }
    }
#endif

#if (1 == CHK_INTER_LAYER)
    for(i = 0; i < NUM_LAYER - 1; i++)
    {
        check_intermediate_layer_results(i);
    }
#endif

    memcpy(sa_image_in_1_u08, sa_image_in_0_u08, WID_SRC * HEI_SRC * CHN_SRC * sizeof(unsigned char));
    memset(sa_out_f32, 0, WID_DST * HEI_DST * CHN_DST * sizeof(float));

#if 1
    clk_srt = clock();
    for(i = 0; i < 10; i++)
    {
        yolo_main(sa_out_f32, sa_image_in_1_u08);
    }
    clk_end = clock();
    printf("yolo 2 10 times: %f s\n", (double)(clk_end - clk_srt) / CLOCKS_PER_SEC);
#endif

#if 0
    for(k = 0; k < CHN_DST; k++)
    {
        for(j = 0; j < HEI_DST; j++)
        {
            for(i = 0; i < WID_DST; i++)
            {
                if(fabsf(sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST] - spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]) > ACCEPTABLE_DIFF)
                {
                    printf("final results mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST], spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]);
                }
            }
        }
    }
#endif

#if (1 == DEBUG_WRITING)
    fclose(fp_fprintf_debug);
#endif

    if(0 == fread_return)
    {
        printf("problem on fread\n");
    }

    hipFree(sp_gpu_int_0_f32);
    hipFree(sp_gpu_int_1_f32);
    hipFree(sp_gpu_int_16_f32);
    hipFree(sp_gpu_int_24_f32);
    hipFree(sp_gpu_int_27_f32);
    for(i = 0; i < NUM_LAYER; i++)
    {
        free(spa_out_f32[i]);
        if(sa_typ_s32[i] == 0) /* convolutional */
        {
            free(spa_weights_f32[i]);
            hipFree(sp_gpu_weights_f32[i]);
            if(sa_ibn_s32[i] == 1)
            {
                free(spa_mean_f32[i]);
                free(spa_variance_f32[i]);
                free(spa_scales_f32[i]);
                hipFree(sp_gpu_mean_f32[i]);
                hipFree(sp_gpu_variance_f32[i]);
                hipFree(sp_gpu_scales_f32[i]);
            }
            free(spa_biases_f32[i]);
            hipFree(sp_gpu_biases_f32[i]);
        }
        free(spa_ref_f32[i]);
    }

    hipFree(sp_gpu_input_f32);
    hipFree(sp_gpu_workspace_f32);
    hipFree(sp_gpu_image_in_u08);
    hipFree(sp_gpu_image_in_f32);
    hipFree(sp_gpu_resized_f32);
    hipFree(sp_gpu_part_f32);

    return 0;
}

__global__ void u08_to_f32_3ch_kernel(float *p_out_f32, unsigned char *p_src_u08, int wid_s32, int hei_s32)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < wid_s32) && (jCount_s32 < hei_s32) )
    {
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32] / 255.f;
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32] / 255.f;
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32 * 2] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32 * 2] / 255.f;
    }
}

#define W_SCALE (1.263591f)
#define H_SCALE (1.263736f)

__global__ void resize_image_1_kernel(float *part, float *im)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 576) )
    {
        float sx = iCount_s32 * W_SCALE;
        int ix = (int) sx;
        float dx = sx - ix;
        float val;
        val = (1 - dx) * im[0 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[0 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[0 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
        val = (1 - dx) * im[1 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[1 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[1 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
        val = (1 - dx) * im[2 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[2 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[2 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
    }
}

__global__ void resize_image_2_kernel(float *resized, float *part)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 456) )
    {
        float sy = jCount_s32 * H_SCALE;
        int iy = (int) sy;
        float dy = sy - iy;
        float val;
        val = (1-dy) * part[0 * 608 * 576 + iy * 608 + iCount_s32];
        resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[0 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
        val = (1-dy) * part[1 * 608 * 576 + iy * 608 + iCount_s32];
        resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[1 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
        val = (1-dy) * part[2 * 608 * 576 + iy * 608 + iCount_s32];
        resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[2 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
    }
}

__global__ void fill_image_kernel(float *boxed)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < HEI_SIZED) )
    {
        boxed[iCount_s32 + jCount_s32 * WID_SIZED] = 0.5f;
        boxed[iCount_s32 + jCount_s32 * WID_SIZED + WID_SIZED * HEI_SIZED] = 0.5f;
        boxed[iCount_s32 + jCount_s32 * WID_SIZED + WID_SIZED * HEI_SIZED * 2] = 0.5f;
    }
}

__global__ void embed_image_kernel(float *resized, float *boxed, int dx, int dy)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 456) )
    {
        float val;
        val = resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[0 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
        val = resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[1 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
        val = resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[2 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
    }
}

static void yolo_main(float *p_out_f32, unsigned char *p_image_in_u08)
{
#if (1 == CHK_INTER_LAYER)
    int i, j, k;
#endif
    int l;
    dim3 grid_img_resize_0( 16, 16 );
    dim3 grid_numblocks_resize_0( WID_SRC / grid_img_resize_0.x, HEI_SRC / grid_img_resize_0.y );
    dim3 grid_img_resize_1( 16, 16 );
    dim3 grid_numblocks_resize_1( WID_SIZED / grid_img_resize_1.x, HEI_SRC / grid_img_resize_1.y );
    dim3 grid_img_resize_2( 16, 8 );
    dim3 grid_numblocks_resize_2( WID_SIZED / grid_img_resize_2.x, 456 / grid_img_resize_2.y );
    dim3 grid_img_resize_3( 16, 16 );
    dim3 grid_numblocks_resize_3( WID_SIZED / grid_img_resize_3.x, HEI_SIZED / grid_img_resize_3.y );
    dim3 grid_img_resize_4( 16, 8 );
    dim3 grid_numblocks_resize_4( WID_SIZED / grid_img_resize_4.x, 456 / grid_img_resize_4.y );

    hipMemcpy(sp_gpu_image_in_u08, p_image_in_u08, WID_SRC * HEI_SRC * CHN_SRC * sizeof(unsigned char), hipMemcpyHostToDevice);

    u08_to_f32_3ch_kernel<<<grid_numblocks_resize_0, grid_img_resize_0>>>(sp_gpu_image_in_f32, sp_gpu_image_in_u08, WID_SRC, HEI_SRC);
    resize_image_1_kernel<<<grid_numblocks_resize_1, grid_img_resize_1>>>(sp_gpu_part_f32, sp_gpu_image_in_f32);
    resize_image_2_kernel<<<grid_numblocks_resize_2, grid_img_resize_2>>>(sp_gpu_resized_f32, sp_gpu_part_f32);
    fill_image_kernel<<<grid_numblocks_resize_3, grid_img_resize_3>>>(sp_gpu_input_f32);
    embed_image_kernel<<<grid_numblocks_resize_4, grid_img_resize_4>>>(sp_gpu_resized_f32, sp_gpu_input_f32, 0, 76);
    check_error(hipPeekAtLastError());

#if (1 == CHK_INTER_LAYER)
    hipMemcpy(sa_image_sized_f32, sp_gpu_input_f32, WID_SIZED * HEI_SIZED * CHN_SRC * sizeof(float), hipMemcpyDeviceToHost);

    for(k = 0; k < CHN_SRC; k++)
    {
        for(j = 0; j < HEI_SIZED; j++)
        {
            for(i = 0; i < WID_SIZED; i++)
            {
                if(fabsf(sa_image_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED] - sa_ref_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED]) > ACCEPTABLE_DIFF)
                {
                    printf("resize mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_image_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED], sa_ref_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED]);
                }
            }
        }
    }
#endif

    l = 0;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_input_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], CHN_SRC, sa_wid_s32[l], sa_hei_s32[l], WID_SIZED, HEI_SIZED, 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

#if 0
    l = 1;
    forward_maxpool_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 2;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 3;
    forward_maxpool_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);


    l = 4;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 5;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 6;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 7;
    forward_maxpool_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 8;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 9;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 10;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 11;
    forward_maxpool_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 12;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 13;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 14;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 15;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 16;
    forward_convolutional_layer_gpu(sp_gpu_int_16_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 17;
    forward_maxpool_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_16_f32, sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 18;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 19;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 20;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 21;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 22;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 23;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 24;
    forward_convolutional_layer_gpu(sp_gpu_int_24_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 25;
    forward_route_layer_25_gpu(sp_gpu_int_0_f32, sp_gpu_int_16_f32);

    l = 26;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 27;
    forward_reorg_layer_gpu(sp_gpu_int_27_f32, sp_gpu_int_1_f32, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l - 1], 1, 2);

    l = 28;
    forward_route_layer_28_gpu(sp_gpu_int_0_f32, sp_gpu_int_27_f32, sp_gpu_int_24_f32);

    l = 29;
    forward_convolutional_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 30;
    forward_convolutional_layer_gpu(sp_gpu_int_0_f32, sp_gpu_int_1_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LINEAR);

    l = 31;
    forward_region_layer_gpu(sp_gpu_int_1_f32, sp_gpu_int_0_f32, spa_out_f32[l], 1, sa_wid_s32[l - 1] * sa_hei_s32[l - 1] * sa_chn_s32[l - 1], 5, sa_wid_s32[l - 1], sa_hei_s32[l - 1], 4, 0, 80, sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l]);
#endif

    hipMemcpy(p_out_f32, sp_gpu_int_1_f32, WID_DST * HEI_DST * CHN_DST * sizeof(float), hipMemcpyDeviceToHost);
}

static float *cuda_make_array(float *x, size_t n)
{
    float *x_gpu;
    size_t size = sizeof(float)*n;
    hipError_t status = hipMalloc((void **)&x_gpu, size);
    check_error(status);
    if(x){
        status = hipMemcpy(x_gpu, x, size, hipMemcpyHostToDevice);
        check_error(status);
    } else {
        fill_gpu(n, 0, x_gpu, 1);
    }
    if(!x_gpu) printf("Cuda malloc failed\n");
    return x_gpu;
}

static void check_error(hipError_t status)
{
    //hipDeviceSynchronize();
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        printf("CUDA Error: %s\n", s);
    } 
    if (status2 != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        printf("CUDA Error Prev: %s\n", s);
    } 
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

static void fill_gpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

static dim3 cuda_gridsize(size_t n)
{
    size_t k = (n-1) / BLOCK + 1;
    size_t x = k;
    size_t y = 1;
    if(x > 65535){
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    //printf("%ld %ld %ld %ld\n", n, x, y, x*y*BLOCK);
    return d;
}

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

static void im2col_gpu(float *im, int channels, int height, int width, int ksize, int stride, int pad, float *data_col)
{
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}

static void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, float *A_gpu, int lda, float *B_gpu, int ldb, float BETA, float *C_gpu, int ldc)
{
    hipblasHandle_t handle = blas_handle();
    hipblasStatus_t status = hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N), 
            (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
}

__global__ void convolution_kernel(float *p_out_f32, const float *p_in_f32, const float *p_weights_f32, const int chn_in_s32, const int wid_in_s32, const int hei_in_s32, const int chn_out_s32, const int wid_out_s32, const int hei_out_s32, const int ker_s32, const int pad_s32)
{
    //printf("in convolution_kernel: %d, %d, %d, %d, %d, %d, %d, %d\n", chn_in_s32, wid_in_s32, hei_in_s32, chn_out_s32, wid_out_s32, hei_out_s32, ker_s32, pad_s32);

#if 0
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int co = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int i = blockIdx.y * blockDim.y + threadIdx_y_s32;
    int ci, kw, kh, x, y, j;
    float wei_f32;
    float acc_f32;
    __shared__ float gpu_sa_src_f32[(WID_SIZED + 2) * 3];
    __shared__ float gpu_sa_wei_f32[3 * 3 * 3 * 32];

    if(co < chn_out_s32)
    {
        for(ci = 0; ci < chn_in_s32; ci++)
        {
            for(kh = 0; kh < ker_s32; kh++)
            {
                for(kw = 0; kw < ker_s32; kw++)
                {
                    gpu_sa_wei_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw] = p_weights_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw];
                }
            }
        }
    } 

    __syncthreads();

    if(co < chn_out_s32)
    {
        for(ci = 0; ci < chn_in_s32; ci++)
        {
            for(kh = 0; kh < ker_s32; kh++)
            {
                for(kw = 0; kw < ker_s32; kw++)
                {
                    for(j = 0; j < hei_out_s32; j++)
                    {
                        if(i < wid_out_s32)
                        {
#if 0
                            x = (i - pad_s32) + kw;
                            y = (j - pad_s32) + kh;

                            if((x >= 0) && (x < wid_in_s32) && (y >= 0) && (y < hei_in_s32))
                            {
                                gpu_sa_src_f32[i] = p_in_f32[ci * wid_in_s32 * hei_in_s32 + y * wid_in_s32 + x];
                            }
                            else
                            {
                                gpu_sa_src_f32[i] = 0.f;
                            }

                            __syncthreads();
#endif

                            x = (i - pad_s32) + kw;
                            y = (j - pad_s32) + kh;

                            wei_f32 = gpu_sa_wei_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw];

                            acc_f32 = p_out_f32[co * wid_out_s32 * hei_out_s32 + j * wid_out_s32 + i];

                            if((x >= 0) && (x < wid_in_s32) && (y >= 0) && (y < hei_in_s32))
                            {
                                acc_f32 += p_in_f32[ci * wid_in_s32 * hei_in_s32 + (j - pad_s32 + kh) * wid_in_s32 + (i - pad_s32) + kw] * wei_f32;
                            }
                            
                            p_out_f32[co * wid_out_s32 * hei_out_s32 + j * wid_out_s32 + i] = acc_f32;
                        }
                    }
                }
            }
        }
    } 
#else
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int j = blockIdx.y * blockDim.y + threadIdx_y_s32;
    int ci, co, kw, kh, x, y;
    float src_f32;
    float wei_f32;
    float acc_f32;
    //__shared__ float gpu_sa_src_f32[3 * 3 * 3];
    __shared__ float gpu_sa_wei_f32[3 * 3 * 3 * 32];

#if 1
    for(co = 0; co < chn_out_s32; co++)
    {
        for(ci = 0; ci < chn_in_s32; ci++)
        {
            for(kh = 0; kh < ker_s32; kh++)
            {
                for(kw = 0; kw < ker_s32; kw++)
                {
                    gpu_sa_wei_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw] = p_weights_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw];
                }
            }
        }
    } 

    __syncthreads();
#endif

    for(co = 0; co < chn_out_s32; co++)
    {
        if(j < hei_out_s32)
        {
            if(i < wid_out_s32)
            {
                acc_f32 = 0.0f;

                for(ci = 0; ci < chn_in_s32; ci++)
                {
                    for(kh = 0; kh < ker_s32; kh++)
                    {
                        for(kw = 0; kw < ker_s32; kw++)
                        {
                            x = (i - pad_s32) + kw;
                            y = (j - pad_s32) + kh;

                            if((x >= 0) && (x < wid_in_s32) && (y >= 0) && (y < hei_in_s32))
                            {
                                src_f32 = p_in_f32[ci * wid_in_s32 * hei_in_s32 + (j - pad_s32 + kh) * wid_in_s32 + (i - pad_s32) + kw];
                                wei_f32 = gpu_sa_wei_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw];
                                acc_f32 += src_f32 * wei_f32;
#if (1 == DEBUG_WRITING)
                                if((co == 0) && (ci == 0) && ((j < 20) || ((j > 60) && (j < 80))))
                                {
                                    printf("kw: %d, kh: %d, ci: %d, i: %d, j: %d, in: %f, wei: %f, acc: %f\n", kw, kh, ci, i, j, src_f32, wei_f32, acc_f32);
                                }
#endif
                            }
                        }
                    }
                }

                __syncthreads();

                p_out_f32[co * wid_out_s32 * hei_out_s32 + j * wid_out_s32 + i] = acc_f32;

#if (1 == DEBUG_WRITING)
                if((co == 0) && ((j < 20) || ((j > 60) && (j < 80))))
                {
                    printf("i: %d, j: %d, out: %f\n", i, j, acc_f32);
                }
#endif
            }
        }
    } 
#endif
}

static void forward_convolutional_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_weights_gpu, float *workspace_gpu, float *mean_gpu, float *variance_gpu, float *scales_gpu, float *biases_gpu, int l_outputs, int l_n, int l_size, int l_c, int l_out_w, int l_out_h, int l_w, int l_h, int l_stride, int l_pad, int l_batch_normalize, ACTIVATION l_activation)
{
#if 1
    fill_gpu(l_outputs, 0, l_output_gpu, 1);

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l_srcTensorDesc,
                net.input_gpu,
                l_weightDesc,
                l_weights_gpu,
                l_convDesc,
                l_fw_algo,
                net.workspace,
                l_workspace_size,
                &one,
                l_dstTensorDesc,
                l_output_gpu);

#else
    int m = l_n;
    int k = l_size*l_size*l_c;
    int n = l_out_w*l_out_h;
    float *a = l_weights_gpu;
    float *b = workspace_gpu;
    float *c = l_output_gpu;

    im2col_gpu(input_gpu,
        l_c, l_h, l_w, l_size, l_stride, l_pad, b);
    gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
#endif
#endif

#if 0
    {
        static float sa_src_f32[WID_SIZED * HEI_SIZED * CHN_SRC];
        static float sa_dst_f32[WID_SIZED * HEI_SIZED * 32] = { 0.f, };
        static float sa_wei_f32[3 * 3 * 3 * 32];
        static float sa_ref_f32[WID_SIZED * HEI_SIZED * 32];
        int i, j, k;
#if 0
        dim3 threadsperblock(2048); // 2048
        dim3 numblocks(32); // 32
#else
        dim3 threadsperblock(32, 32); // 2048
        dim3 numblocks(19, 19); // 32
#endif

#if 0
        hipMemcpy(sa_ref_f32, l_output_gpu, l_out_w * l_out_h * 32 * sizeof(float), hipMemcpyDeviceToHost);
        fill_gpu(l_outputs, 0, l_output_gpu, 1);
#endif

#if 0
        hipMemcpy(sa_src_f32, input_gpu, WID_SIZED * HEI_SIZED * CHN_SRC * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(sa_wei_f32, l_weights_gpu, l_size * l_size * l_c * 32 * sizeof(float), hipMemcpyDeviceToHost);

        convolution_ref_c(sa_dst_f32, sa_src_f32, sa_wei_f32, l_c, l_w, l_h, 32, l_out_w, l_out_h, l_size, l_pad);
#else
        convolution_kernel<<<threadsperblock, numblocks>>>(l_output_gpu, input_gpu, l_weights_gpu, l_c, l_w, l_h, 32, l_out_w, l_out_h, l_size, l_pad);

        //hipMemcpy(sa_dst_f32, l_output_gpu, l_out_w * l_out_h * 32 * sizeof(float), hipMemcpyDeviceToHost);
#endif

#if 0
        for(k = 0; k < 32; k++)
        {
            for(j = 0; j < l_out_h; j++)
            {
                for(i = 0; i < l_out_w; i++)
                {
                    if(fabsf(sa_dst_f32[i + j * l_out_w + k * l_out_w * l_out_h] - sa_ref_f32[i + j * l_out_w + k * l_out_w * l_out_h]) > ACCEPTABLE_DIFF)
                    {
                        printf("mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_dst_f32[i + j * l_out_w + k * l_out_w * l_out_h] , sa_ref_f32[i + j * l_out_w + k * l_out_w * l_out_h]);
                    }
                }
            }
        }
#endif
    }
#endif

    if (l_batch_normalize) {
        normalize_gpu(l_output_gpu, mean_gpu, variance_gpu, 1, l_n, l_out_w * l_out_h);
        scale_bias_gpu(l_output_gpu, scales_gpu, 1, l_n, l_out_w * l_out_h);
        add_bias_gpu(l_output_gpu, biases_gpu, 1, l_n, l_out_w * l_out_h);
    } else {
        add_bias_gpu(l_output_gpu, biases_gpu, 1, l_n, l_out_w * l_out_h);
    }

    activate_array_gpu(l_output_gpu, l_outputs, l_activation);
}

static void convolution_ref_c(float * __restrict p_out_f32, const float * __restrict p_in_f32, const float * __restrict p_weights_f32, const int chn_in_s32, const int wid_in_s32, const int hei_in_s32, const int chn_out_s32, const int wid_out_s32, const int hei_out_s32, const int ker_s32, const int pad_s32)
{
    int i, j, ci, co, kw, kh, x, y;
    float src_f32;
    float wei_f32;
    float acc_f32;

    for(co = 0; co < chn_out_s32; co++)
    {
        for(ci = 0; ci < chn_in_s32; ci++)
        {
            for(kh = 0; kh < ker_s32; kh++)
            {
                for(kw = 0; kw < ker_s32; kw++)
                {
                    for(j = 0; j < hei_out_s32; j++)
                    {
                        for(i = 0; i < wid_out_s32; i++)
                        {
                            acc_f32 = p_out_f32[co * wid_out_s32 * hei_out_s32 + j * wid_out_s32 + i];

                            x = (i - pad_s32) + kw;
                            y = (j - pad_s32) + kh;

                            if((x < 0) || (x >= wid_in_s32) || (y < 0) || (y >= hei_in_s32))
                            {
                                src_f32 = 0.f;
                            }
                            else
                            {
                                src_f32 = p_in_f32[ci * wid_in_s32 * hei_in_s32 + (j - pad_s32 + kh) * wid_in_s32 + (i - pad_s32) + kw];
                            }
                            wei_f32 = p_weights_f32[co * ker_s32 * ker_s32 * chn_in_s32 + ci * ker_s32 * ker_s32 + kh * ker_s32 + kw];
                            acc_f32 += src_f32 * wei_f32;
#if (1 == DEBUG_WRITING)
                            if((co == 0) && (ci == 0) && ((j < 20) || ((j > 60) && (j < 80))))
                            {
                                fprintf(fp_fprintf_debug, "kw: %d, kh: %d, ci: %d, i: %d, j: %d, in: %f, wei: %f, acc: %f\n", kw, kh, ci, i, j, src_f32, wei_f32, acc_f32);
                            }
#endif

                            p_out_f32[co * wid_out_s32 * hei_out_s32 + j * wid_out_s32 + i] = acc_f32;
                        }
                    }
                }
            }
        }
    } 
}

static hipblasHandle_t blas_handle()
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int i = cuda_get_device();
    if(!init[i]) {
        hipblasCreate(&handle[i]);
        init[i] = 1;
    }
    return handle[i];
}

static int cuda_get_device()
{
    int n = 0;
    hipError_t status = hipGetDevice(&n);
    check_error(status);
    return n;
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrtf(variance[f] + .00001f));
}

static void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

static void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

static void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}

__device__ float lhtan_activate_kernel(float x)
{
    if(x < 0) return .001f*x;
    if(x > 1) return .001f*(x-1.f) + 1.f;
    return x;
}
__device__ float lhtan_gradient_kernel(float x)
{
    if(x > 0 && x < 1) return 1;
    return .001;
}

__device__ float hardtan_activate_kernel(float x)
{
    if (x < -1) return -1;
    if (x > 1) return 1;
    return x;
}
__device__ float linear_activate_kernel(float x){return x;}
__device__ float logistic_activate_kernel(float x){return 1.f/(1.f + expf(-x));}
__device__ float loggy_activate_kernel(float x){return 2.f/(1.f + expf(-x)) - 1;}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float elu_activate_kernel(float x){return (x >= 0)*x + (x < 0)*(expf(x)-1);}
__device__ float relie_activate_kernel(float x){return (x>0) ? x : .01f*x;}
__device__ float ramp_activate_kernel(float x){return x*(x>0)+.1f*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1f*x;}
__device__ float tanh_activate_kernel(float x){return (2.f/(1 + expf(-2*x)) - 1);}
__device__ float plse_activate_kernel(float x)
{
    if(x < -4) return .01f * (x + 4);
    if(x > 4)  return .01f * (x - 4) + 1;
    return .125f*x + .5f;
}
__device__ float stair_activate_kernel(float x)
{
    int n = floorf(x);
    if (n%2 == 0) return floorf(x/2);
    else return (x - n) + floorf(x/2);
}

__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_activate_kernel(x);
        case LOGISTIC:
            return logistic_activate_kernel(x);
        case LOGGY:
            return loggy_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case ELU:
            return elu_activate_kernel(x);
        case RELIE:
            return relie_activate_kernel(x);
        case RAMP:
            return ramp_activate_kernel(x);
        case LEAKY:
            return leaky_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
        case PLSE:
            return plse_activate_kernel(x);
        case STAIR:
            return stair_activate_kernel(x);
        case HARDTAN:
            return hardtan_activate_kernel(x);
        case LHTAN:
            return lhtan_activate_kernel(x);
    }
    return 0;
}

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = activate_kernel(x[i], a);
}

static void activate_array_gpu(float *x, int n, ACTIVATION a) 
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
    check_error(hipPeekAtLastError());
}

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
}

static void forward_maxpool_layer_gpu(float *l_output_gpu, float *input_gpu, int layer_out_w, int layer_out_h, int layer_batch, int layer_w, int layer_h, int layer_c, int layer_stride, int layer_size, int layer_pad)
{
    int h = layer_out_h;
    int w = layer_out_w;
    int c = layer_c;

    size_t n = h*w*c*layer_batch;

    forward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer_h, layer_w, layer_c, layer_stride, layer_size, layer_pad, input_gpu, l_output_gpu);
    check_error(hipPeekAtLastError());
}

static void forward_route_layer_25_gpu(float *l_output_gpu, float *input_l16)
{
    int offset = 0;
    float *input = input_l16;
    int input_size = 739328;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
}

static void forward_route_layer_28_gpu(float *l_output_gpu, float *input_l27, float *input_l24)
{
    int offset = 0;
    float *input = input_l27;
    int input_size = 92416;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
    offset += input_size;
    input = input_l24;
    input_size = 369664;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
}

static void copy_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

static void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

static void forward_reorg_layer_gpu(float *l_output_gpu, float *input_gpu, int l_w, int l_h, int l_c, int l_batch, int l_stride)
{
    reorg_gpu(input_gpu, l_w, l_h, l_c, l_batch, l_stride, 0, l_output_gpu);
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
}

static void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int size = w*h*c*batch;
    reorg_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, w, h, c, batch, stride, forward, out);
    check_error(hipPeekAtLastError());
}

static void forward_region_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_output, int l_batch, int l_inputs, int l_n, int l_w, int l_h, int l_coords, int l_background, int l_classes, int l_outputs)
{
    copy_gpu(l_batch*l_inputs, input_gpu, 1, l_output_gpu, 1);
    int b, n;
    for (b = 0; b < l_batch; ++b){
        for(n = 0; n < l_n; ++n){
            int index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, 0);
            activate_array_gpu(l_output_gpu + index, 2*l_w*l_h, LOGISTIC);
            index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, l_coords);
            activate_array_gpu(l_output_gpu + index,   l_w*l_h, LOGISTIC);
            index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, l_coords + 1);
        }
    }
    {
        int index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, 0, 0, l_coords + !l_background);
        softmax_gpu(input_gpu + index, l_classes + l_background, l_batch*l_n, l_inputs/l_n, l_w*l_h, 1, l_w*l_h, 1, l_output_gpu + index);
    }
    {
        cuda_pull_array(l_output_gpu, l_output, l_batch*l_outputs);
    }
}

static int entry_index(int l_w, int l_h, int l_outputs, int l_coords, int l_classes, int batch, int location, int entry)
{
    int n =   location / (l_w*l_h);
    int loc = location % (l_w*l_h);
    return batch*l_outputs + n*l_w*l_h*(l_coords+l_classes+1) + entry*l_w*l_h + loc;
}

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i*stride];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch*groups) return;
    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}

static void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    softmax_kernel<<<cuda_gridsize(batch*groups), BLOCK>>>(input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
    check_error(hipPeekAtLastError());
}

static void cuda_pull_array(float *x_gpu, float *x, size_t n)
{
    size_t size = sizeof(float)*n;
    hipError_t status = hipMemcpy(x, x_gpu, size, hipMemcpyDeviceToHost);
    check_error(status);
}

#if (1 == CHK_INTER_LAYER)
static void check_intermediate_layer_results(int l)
{
    int i, j, k;

    hipMemcpy(spa_out_f32[l], sp_gpu_out_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l] * sizeof(float), hipMemcpyDeviceToHost);
    for(k = 0; k < sa_chn_s32[l]; k++)
    {
        for(j = 0; j < sa_hei_s32[l]; j++)
        {
            for(i = 0; i < sa_wid_s32[l]; i++)
            {
                if(fabsf(spa_out_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]] - spa_ref_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]]) > ACCEPTABLE_DIFF)
                {
                    printf("layer %d mismatch: w %d, h %d, c %d, out %f, GT %f\n", l, i, j, k, spa_out_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]], spa_ref_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]]);
                }
            }
        }
    }
}
#endif

