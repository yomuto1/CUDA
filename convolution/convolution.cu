#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#define DEBUG_WRITING (0)

#define WID_SRC (768)
#define HEI_SRC (576)
#define CHN_SRC (3)
#define WID_SIZED (608)
#define HEI_SIZED (608)
#define WID_DST (19)
#define HEI_DST (19)
#define CHN_DST (425)

#define SIZE_MAX_WORKSPACE (30000000)
#define NUM_LAYER (32)

#define BLOCK (512)

#define CHK_INTER_LAYER (0)
#define ACCEPTABLE_DIFF (0.005f)

#if (1 == DEBUG_WRITING)
FILE *fp_fprintf_debug;
#endif

typedef struct {
    int w;
    int h;
    int c;
    float *data;
} image;

typedef enum{
    LOGISTIC, RELU, RELIE, LINEAR, RAMP, TANH, PLSE, LEAKY, ELU, LOGGY, STAIR, HARDTAN, LHTAN
} ACTIVATION;

static int sa_typ_s32[NUM_LAYER];
static int sa_wid_s32[NUM_LAYER];
static int sa_hei_s32[NUM_LAYER];
static int sa_chn_s32[NUM_LAYER];
static int sa_ker_s32[NUM_LAYER];
static int sa_pad_s32[NUM_LAYER];
static int sa_ibn_s32[NUM_LAYER];
static int sa_nwe_s32[NUM_LAYER];
static unsigned char sa_image_in_u08[WID_SRC * HEI_SRC * CHN_SRC];
#if (1 == CHK_INTER_LAYER)
static float sa_image_sized_f32[WID_SIZED * HEI_SIZED * CHN_SRC];
#endif
static float sa_tmp_buf_f32[SIZE_MAX_WORKSPACE];
static float sa_out_f32[WID_DST * HEI_DST * CHN_DST];
static float *spa_out_f32[NUM_LAYER];
static float *spa_weights_f32[NUM_LAYER];
static float *spa_mean_f32[NUM_LAYER];
static float *spa_variance_f32[NUM_LAYER];
static float *spa_scales_f32[NUM_LAYER];
static float *spa_biases_f32[NUM_LAYER];
static float *sp_gpu_out_f32[NUM_LAYER];
static float *sp_gpu_weights_f32[NUM_LAYER];
static float *sp_gpu_mean_f32[NUM_LAYER];
static float *sp_gpu_variance_f32[NUM_LAYER];
static float *sp_gpu_scales_f32[NUM_LAYER];
static float *sp_gpu_biases_f32[NUM_LAYER];
static float *sp_gpu_input_f32;
static float *sp_gpu_workspace_f32;
static float sa_ref_sized_f32[WID_SIZED * HEI_SIZED * CHN_SRC];
static float *spa_ref_f32[NUM_LAYER];

static void yolo_main(float *p_out_f32, unsigned char *p_image_in_u08);
static float *cuda_make_array(float *x, size_t n);
static void check_error(hipError_t status);
static void fill_gpu(int N, float ALPHA, float * X, int INCX);
static dim3 cuda_gridsize(size_t n);
static void im2col_gpu(float *im, int channels, int height, int width, int ksize, int stride, int pad, float *data_col);
static void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, float *A_gpu, int lda, float *B_gpu, int ldb, float BETA, float *C_gpu, int ldc);
static void forward_convolutional_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_weights_gpu, float *workspace_gpu, float *mean_gpu, float *variance_gpu, float *scales_gpu, float *biases_gpu, int l_outputs, int l_n, int l_size, int l_c, int l_out_w, int l_out_h, int l_w, int l_h, int l_stride, int l_pad, int l_batch_normalize, ACTIVATION l_activation);
static hipblasHandle_t blas_handle();
static int cuda_get_device();
static void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial);
static void scale_bias_gpu(float *output, float *biases, int batch, int n, int size);
static void add_bias_gpu(float *output, float *biases, int batch, int n, int size);
static void activate_array_gpu(float *x, int n, ACTIVATION a);
static void forward_maxpool_layer_gpu(float *l_output_gpu, float *input_gpu, int layer_out_w, int layer_out_h, int layer_batch, int layer_w, int layer_h, int layer_c, int layer_stride, int layer_size, int layer_pad);
static void forward_route_layer_25_gpu(float *l_output_gpu, float *input_l16);
static void forward_route_layer_28_gpu(float *l_output_gpu, float *input_l27, float *input_l24);
static void copy_gpu(int N, float * X, int INCX, float * Y, int INCY);
static void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY);
static void forward_reorg_layer_gpu(float *l_output_gpu, float *input_gpu, int l_w, int l_h, int l_c, int l_batch, int l_stride);
static void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out);
static void forward_region_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_output, int l_batch, int l_inputs, int l_n, int l_w, int l_h, int l_coords, int l_background, int l_classes, int l_outputs);
static int entry_index(int l_w, int l_h, int l_outputs, int l_coords, int l_classes, int batch, int location, int entry);
static void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output);
static void cuda_pull_array(float *x_gpu, float *x, size_t n);
#if (1 == CHK_INTER_LAYER)
static void check_intermediate_layer_results(int l);
#endif

static unsigned char *sp_gpu_image_in_u08;
static float *sp_gpu_image_in_f32;
static float *sp_gpu_resized_f32;
static float *sp_gpu_part_f32;

int main(void)
{
    FILE *fp;
    FILE *fp_results;
    FILE *fp_weights;
    FILE *fp_mean;
    FILE *fp_variance;
    FILE *fp_scales;
    FILE *fp_biases;
    FILE *fp_netinfo;
    int i, j, k;
    size_t fread_return;
    clock_t clk_srt, clk_end;

    printf("yolo reference CUDA code by Hyuk Lee\n");

#if (1 == DEBUG_WRITING)
    fp_fprintf_debug = fopen("ref_c_debug.txt", "w");
#endif

    /* read net info */
    fp_netinfo = fopen("yolo_gpu_netinfo.bin", "rb");
    if(NULL == fp_netinfo)
    {
        printf("yolo_gpu_netinfo fopen error\n");
        return -1;
    }
    /* load weights */
    fp_weights = fopen("yolo_gpu_weights.bin", "rb");
    if(NULL == fp_weights)
    {
        printf("yolo_gpu_weights fopen error\n");
        return -1;
    }
    fp_mean = fopen("yolo_gpu_mean.bin", "rb");
    if(NULL == fp_mean)
    {
        printf("yolo_gpu_mean fopen error\n");
        return -1;
    }
    fp_variance = fopen("yolo_gpu_variance.bin", "rb");
    if(NULL == fp_variance)
    {
        printf("yolo_gpu_variance fopen error\n");
        return -1;
    }
    fp_scales = fopen("yolo_gpu_scales.bin", "rb");
    if(NULL == fp_scales)
    {
        printf("yolo_gpu_scales fopen error\n");
        return -1;
    }
    fp_biases = fopen("yolo_gpu_biases.bin", "rb");
    if(NULL == fp_biases)
    {
        printf("yolo_gpu_biases fopen error\n");
        return -1;
    }
    /* load ref data */
    fp = fopen("yolo_image_sized.bin", "rb");
    if(NULL == fp)
    {
        printf("yolo_image_sized fopen error\n");
        return -1;
    }
    fread_return = fread(sa_ref_sized_f32, WID_SIZED * HEI_SIZED * CHN_SRC, sizeof(float), fp);
    fclose(fp);
    fp_results = fopen("yolo_gpu_intermediate_results.bin", "rb");
    if(NULL == fp_results)
    {
        printf("yolo_gpu_results fopen error\n");
        return -1;
    }
    for(i = 0; i < NUM_LAYER; i++)
    {
        fread_return = fread(&sa_typ_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_wid_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_hei_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_chn_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_ker_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_pad_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_ibn_s32[i], 1, sizeof(int), fp_netinfo);        
        fread_return = fread(&sa_nwe_s32[i], 1, sizeof(int), fp_netinfo);        
        spa_out_f32[i] = (float *)malloc(sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i] * sizeof(float));
        sp_gpu_out_f32[i] = cuda_make_array(spa_out_f32[i], sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i]);
        if(sa_typ_s32[i] == 0) /* convolutional */
        {
            spa_weights_f32[i] = (float *)malloc(sa_nwe_s32[i] * sizeof(float));
            fread_return = fread(spa_weights_f32[i], sa_nwe_s32[i], sizeof(float), fp_weights);
            sp_gpu_weights_f32[i] = cuda_make_array(spa_weights_f32[i], sa_nwe_s32[i]);
            if(sa_ibn_s32[i] == 1)
            {
                spa_mean_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                spa_variance_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                spa_scales_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
                fread_return = fread(spa_mean_f32[i], sa_chn_s32[i], sizeof(float), fp_mean);
                fread_return = fread(spa_variance_f32[i], sa_chn_s32[i], sizeof(float), fp_variance);
                fread_return = fread(spa_scales_f32[i], sa_chn_s32[i], sizeof(float), fp_scales);
                sp_gpu_mean_f32[i] = cuda_make_array(spa_mean_f32[i], sa_chn_s32[i]);
                sp_gpu_variance_f32[i] = cuda_make_array(spa_variance_f32[i], sa_chn_s32[i]);
                sp_gpu_scales_f32[i] = cuda_make_array(spa_scales_f32[i], sa_chn_s32[i]);
            }
            spa_biases_f32[i] = (float *)malloc(sa_chn_s32[i] * sizeof(float));
            fread_return = fread(spa_biases_f32[i], sa_chn_s32[i], sizeof(float), fp_biases);
            sp_gpu_biases_f32[i] = cuda_make_array(spa_biases_f32[i], sa_chn_s32[i]);
        }
        spa_ref_f32[i] = (float *)malloc(sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i] * sizeof(float));
        fread_return = fread(spa_ref_f32[i], sa_wid_s32[i] * sa_hei_s32[i] * sa_chn_s32[i], sizeof(float), fp_results);
    }
    fclose(fp_results);
    fclose(fp_weights);
    fclose(fp_mean);
    fclose(fp_variance);
    fclose(fp_scales);
    fclose(fp_biases);
    fclose(fp_netinfo);

    /* read input data */
    fp = fopen("yolo_image_in.bin", "rb");
    if(NULL == fp)
    {
        printf("yolo_image_in fopen error\n");
        return -1;
    }
    fread_return = fread(sa_image_in_u08, WID_SRC * HEI_SRC * CHN_SRC, sizeof(unsigned char), fp);
    fclose(fp);

    sp_gpu_workspace_f32 = cuda_make_array(sa_tmp_buf_f32, SIZE_MAX_WORKSPACE);
    sp_gpu_input_f32 = cuda_make_array(sa_tmp_buf_f32, WID_SIZED * HEI_SIZED * CHN_SRC);
    hipMalloc((void **)&sp_gpu_image_in_u08, WID_SRC * HEI_SRC * CHN_SRC * sizeof(unsigned char));
    hipMalloc((void **)&sp_gpu_image_in_f32, WID_SRC * HEI_SRC * CHN_SRC * sizeof(float));
    hipMalloc((void **)&sp_gpu_resized_f32, 608 * 456 * 3 * sizeof(float));
    hipMalloc((void **)&sp_gpu_part_f32, 608 * 576 * 3 * sizeof(float));

    clk_srt = clock();
    yolo_main(sa_out_f32, sa_image_in_u08);
    clk_end = clock();
    printf("yolo 1: %f s\n", (double)(clk_end - clk_srt) / CLOCKS_PER_SEC);

    for(k = 0; k < CHN_DST; k++)
    {
        for(j = 0; j < HEI_DST; j++)
        {
            for(i = 0; i < WID_DST; i++)
            {
                if(fabsf(sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST] - spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]) > ACCEPTABLE_DIFF)
                {
                    printf("final results mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST], spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]);
                }
            }
        }
    }

#if (1 == CHK_INTER_LAYER)
    for(i = 0; i < NUM_LAYER - 1; i++)
    {
        check_intermediate_layer_results(i);
    }
#endif

    clk_srt = clock();
    yolo_main(sa_out_f32, sa_image_in_u08);
    clk_end = clock();
    printf("yolo 2: %f s\n", (double)(clk_end - clk_srt) / CLOCKS_PER_SEC);

    for(k = 0; k < CHN_DST; k++)
    {
        for(j = 0; j < HEI_DST; j++)
        {
            for(i = 0; i < WID_DST; i++)
            {
                if(fabsf(sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST] - spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]) > ACCEPTABLE_DIFF)
                {
                    printf("final results mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_out_f32[i + j * WID_DST + k * WID_DST * HEI_DST], spa_ref_f32[NUM_LAYER - 1][i + j * WID_DST + k * WID_DST * HEI_DST]);
                }
            }
        }
    }

#if (1 == DEBUG_WRITING)
    fclose(fp_fprintf_debug);
#endif

    if(0 == fread_return)
    {
        printf("problem on fread\n");
    }

    for(i = 0; i < NUM_LAYER; i++)
    {
        free(spa_out_f32[i]);
        hipFree(sp_gpu_out_f32[i]);
        if(sa_typ_s32[i] == 0) /* convolutional */
        {
            free(spa_weights_f32[i]);
            hipFree(sp_gpu_weights_f32[i]);
            if(sa_ibn_s32[i] == 1)
            {
                free(spa_mean_f32[i]);
                free(spa_variance_f32[i]);
                free(spa_scales_f32[i]);
                hipFree(sp_gpu_mean_f32[i]);
                hipFree(sp_gpu_variance_f32[i]);
                hipFree(sp_gpu_scales_f32[i]);
            }
            free(spa_biases_f32[i]);
            hipFree(sp_gpu_biases_f32[i]);
        }
        free(spa_ref_f32[i]);
    }

    hipFree(sp_gpu_input_f32);
    hipFree(sp_gpu_workspace_f32);
    hipFree(sp_gpu_image_in_u08);
    hipFree(sp_gpu_image_in_f32);
    hipFree(sp_gpu_resized_f32);
    hipFree(sp_gpu_part_f32);

    return 0;
}

__global__ void u08_to_f32_3ch_kernel(float *p_out_f32, unsigned char *p_src_u08, int wid_s32, int hei_s32)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < wid_s32) && (jCount_s32 < hei_s32) )
    {
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32] / 255.f;
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32] / 255.f;
        p_out_f32[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32 * 2] = p_src_u08[iCount_s32 + jCount_s32 * wid_s32 + wid_s32 * hei_s32 * 2] / 255.f;
    }
}

#define W_SCALE (1.263591f)
#define H_SCALE (1.263736f)

__global__ void resize_image_1_kernel(float *part, float *im)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 576) )
    {
        float sx = iCount_s32 * W_SCALE;
        int ix = (int) sx;
        float dx = sx - ix;
        float val;
        val = (1 - dx) * im[0 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[0 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[0 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
        val = (1 - dx) * im[1 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[1 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[1 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
        val = (1 - dx) * im[2 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix] + dx * im[2 * HEI_SRC * WID_SRC + jCount_s32 * WID_SRC + ix + 1];
        part[2 * WID_SIZED * HEI_SRC + jCount_s32 * WID_SIZED + iCount_s32] = val;
    }
}

__global__ void resize_image_2_kernel(float *resized, float *part)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 456) )
    {
        float sy = jCount_s32 * H_SCALE;
        int iy = (int) sy;
        float dy = sy - iy;
        float val;
        val = (1-dy) * part[0 * 608 * 576 + iy * 608 + iCount_s32];
        resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[0 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
        val = (1-dy) * part[1 * 608 * 576 + iy * 608 + iCount_s32];
        resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[1 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
        val = (1-dy) * part[2 * 608 * 576 + iy * 608 + iCount_s32];
        resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32] = val;
        val = dy * part[2 * 608 * 576 + (iy + 1) * 608 + iCount_s32];
        resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32] += val;
    }
}

__global__ void fill_image_kernel(float *boxed)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < HEI_SIZED) )
    {
        boxed[iCount_s32 + jCount_s32 * WID_SIZED] = 0.5f;
        boxed[iCount_s32 + jCount_s32 * WID_SIZED + WID_SIZED * HEI_SIZED] = 0.5f;
        boxed[iCount_s32 + jCount_s32 * WID_SIZED + WID_SIZED * HEI_SIZED * 2] = 0.5f;
    }
}

__global__ void embed_image_kernel(float *resized, float *boxed, int dx, int dy)
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < WID_SIZED) && (jCount_s32 < 456) )
    {
        float val;
        val = resized[0 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[0 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
        val = resized[1 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[1 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
        val = resized[2 * 608 * 456 + jCount_s32 * 608 + iCount_s32];
        boxed[2 * 608 * 608 + (dy + jCount_s32) * 608 + (dx + iCount_s32)] = val;
    }
}

static void yolo_main(float *p_out_f32, unsigned char *p_image_in_u08)
{
#if (1 == CHK_INTER_LAYER)
    int i, j, k;
#endif
    int l;
    dim3 grid_img_resize_0( 16, 16 );
    dim3 grid_numblocks_resize_0( WID_SRC / grid_img_resize_0.x, HEI_SRC / grid_img_resize_0.y );
    dim3 grid_img_resize_1( 16, 16 );
    dim3 grid_numblocks_resize_1( WID_SIZED / grid_img_resize_1.x, HEI_SRC / grid_img_resize_1.y );
    dim3 grid_img_resize_2( 16, 8 );
    dim3 grid_numblocks_resize_2( WID_SIZED / grid_img_resize_2.x, 456 / grid_img_resize_2.y );
    dim3 grid_img_resize_3( 16, 16 );
    dim3 grid_numblocks_resize_3( WID_SIZED / grid_img_resize_3.x, HEI_SIZED / grid_img_resize_3.y );
    dim3 grid_img_resize_4( 16, 8 );
    dim3 grid_numblocks_resize_4( WID_SIZED / grid_img_resize_4.x, 456 / grid_img_resize_4.y );

    hipMemcpy(sp_gpu_image_in_u08, p_image_in_u08, WID_SRC * HEI_SRC * CHN_SRC * sizeof(unsigned char), hipMemcpyHostToDevice);

    u08_to_f32_3ch_kernel<<<grid_numblocks_resize_0, grid_img_resize_0>>>(sp_gpu_image_in_f32, sp_gpu_image_in_u08, WID_SRC, HEI_SRC);
    resize_image_1_kernel<<<grid_numblocks_resize_1, grid_img_resize_1>>>(sp_gpu_part_f32, sp_gpu_image_in_f32);
    resize_image_2_kernel<<<grid_numblocks_resize_2, grid_img_resize_2>>>(sp_gpu_resized_f32, sp_gpu_part_f32);
    fill_image_kernel<<<grid_numblocks_resize_3, grid_img_resize_3>>>(sp_gpu_input_f32);
    embed_image_kernel<<<grid_numblocks_resize_4, grid_img_resize_4>>>(sp_gpu_resized_f32, sp_gpu_input_f32, 0, 76);
    check_error(hipPeekAtLastError());

#if (1 == CHK_INTER_LAYER)
    cudaMemcpy(sa_image_sized_f32, sp_gpu_input_f32, WID_SIZED * HEI_SIZED * CHN_SRC * sizeof(float), cudaMemcpyDeviceToHost);

    for(k = 0; k < CHN_SRC; k++)
    {
        for(j = 0; j < HEI_SIZED; j++)
        {
            for(i = 0; i < WID_SIZED; i++)
            {
                if(fabsf(sa_image_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED] - sa_ref_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED]) > ACCEPTABLE_DIFF)
                {
                    printf("resize mismatch: w %d, h %d, c %d, out %f, GT %f\n", i, j, k, sa_image_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED], sa_ref_sized_f32[i + j * WID_SIZED + k * WID_SIZED * HEI_SIZED]);
                }
            }
        }
    }
#endif

    l = 0;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_input_f32, sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], CHN_SRC, sa_wid_s32[l], sa_hei_s32[l], WID_SIZED, HEI_SIZED, 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 1;
    forward_maxpool_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 2;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 3;
    forward_maxpool_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);


    l = 4;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 5;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 6;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 7;
    forward_maxpool_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 8;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 9;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 10;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 11;
    forward_maxpool_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 12;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 13;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 14;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 15;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 16;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 17;
    forward_maxpool_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l], sa_hei_s32[l], 1, sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l], 2, 2, 0);

    l = 18;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 19;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 20;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 21;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 22;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 23;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 24;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 25;
    forward_route_layer_25_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[16]);

    l = 26;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 27;
    forward_reorg_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sa_wid_s32[l - 1], sa_hei_s32[l - 1], sa_chn_s32[l - 1], 1, 2);

    l = 28;
    forward_route_layer_28_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[27], sp_gpu_out_f32[24]);

    l = 29;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LEAKY);

    l = 30;
    forward_convolutional_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], sp_gpu_weights_f32[l], sp_gpu_workspace_f32, sp_gpu_mean_f32[l], sp_gpu_variance_f32[l], sp_gpu_scales_f32[l], sp_gpu_biases_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l], sa_chn_s32[l], sa_ker_s32[l], sa_chn_s32[l - 1], sa_wid_s32[l], sa_hei_s32[l], sa_wid_s32[l - 1], sa_hei_s32[l - 1], 1, sa_pad_s32[l], sa_ibn_s32[l], LINEAR);

    l = 31;
    forward_region_layer_gpu(sp_gpu_out_f32[l], sp_gpu_out_f32[l - 1], spa_out_f32[l], 1, sa_wid_s32[l - 1] * sa_hei_s32[l - 1] * sa_chn_s32[l - 1], 5, sa_wid_s32[l - 1], sa_hei_s32[l - 1], 4, 0, 80, sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l]);

    hipMemcpy(p_out_f32, sp_gpu_out_f32[l], WID_DST * HEI_DST * CHN_DST * sizeof(float), hipMemcpyDeviceToHost);
}

static float *cuda_make_array(float *x, size_t n)
{
    float *x_gpu;
    size_t size = sizeof(float)*n;
    hipError_t status = hipMalloc((void **)&x_gpu, size);
    check_error(status);
    if(x){
        status = hipMemcpy(x_gpu, x, size, hipMemcpyHostToDevice);
        check_error(status);
    } else {
        fill_gpu(n, 0, x_gpu, 1);
    }
    if(!x_gpu) printf("Cuda malloc failed\n");
    return x_gpu;
}

static void check_error(hipError_t status)
{
    //cudaDeviceSynchronize();
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        printf("CUDA Error: %s\n", s);
    } 
    if (status2 != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        printf("CUDA Error Prev: %s\n", s);
    } 
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

static void fill_gpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

static dim3 cuda_gridsize(size_t n)
{
    size_t k = (n-1) / BLOCK + 1;
    size_t x = k;
    size_t y = 1;
    if(x > 65535){
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    //printf("%ld %ld %ld %ld\n", n, x, y, x*y*BLOCK);
    return d;
}

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

static void im2col_gpu(float *im, int channels, int height, int width, int ksize, int stride, int pad, float *data_col)
{
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}

static void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, float *A_gpu, int lda, float *B_gpu, int ldb, float BETA, float *C_gpu, int ldc)
{
    hipblasHandle_t handle = blas_handle();
    hipblasStatus_t status = hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N), 
            (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
}

static void forward_convolutional_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_weights_gpu, float *workspace_gpu, float *mean_gpu, float *variance_gpu, float *scales_gpu, float *biases_gpu, int l_outputs, int l_n, int l_size, int l_c, int l_out_w, int l_out_h, int l_w, int l_h, int l_stride, int l_pad, int l_batch_normalize, ACTIVATION l_activation)
{
    fill_gpu(l_outputs, 0, l_output_gpu, 1);

#ifdef CUDNN
    float one = 1;
    cudnnConvolutionForward(cudnn_handle(),
                &one,
                l_srcTensorDesc,
                net.input_gpu,
                l_weightDesc,
                l_weights_gpu,
                l_convDesc,
                l_fw_algo,
                net.workspace,
                l_workspace_size,
                &one,
                l_dstTensorDesc,
                l_output_gpu);

#else
    int m = l_n;
    int k = l_size*l_size*l_c;
    int n = l_out_w*l_out_h;
    float *a = l_weights_gpu;
    float *b = workspace_gpu;
    float *c = l_output_gpu;

    im2col_gpu(input_gpu,
        l_c, l_h, l_w, l_size, l_stride, l_pad, b);
    gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
#endif

    if (l_batch_normalize) {
        normalize_gpu(l_output_gpu, mean_gpu, variance_gpu, 1, l_n, l_out_w * l_out_h);
        scale_bias_gpu(l_output_gpu, scales_gpu, 1, l_n, l_out_w * l_out_h);
        add_bias_gpu(l_output_gpu, biases_gpu, 1, l_n, l_out_w * l_out_h);
    } else {
        add_bias_gpu(l_output_gpu, biases_gpu, 1, l_n, l_out_w * l_out_h);
    }

    activate_array_gpu(l_output_gpu, l_outputs, l_activation);
}

static hipblasHandle_t blas_handle()
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int i = cuda_get_device();
    if(!init[i]) {
        hipblasCreate(&handle[i]);
        init[i] = 1;
    }
    return handle[i];
}

static int cuda_get_device()
{
    int n = 0;
    hipError_t status = hipGetDevice(&n);
    check_error(status);
    return n;
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrtf(variance[f] + .00001f));
}

static void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

static void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

static void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}

__device__ float lhtan_activate_kernel(float x)
{
    if(x < 0) return .001f*x;
    if(x > 1) return .001f*(x-1.f) + 1.f;
    return x;
}
__device__ float lhtan_gradient_kernel(float x)
{
    if(x > 0 && x < 1) return 1;
    return .001;
}

__device__ float hardtan_activate_kernel(float x)
{
    if (x < -1) return -1;
    if (x > 1) return 1;
    return x;
}
__device__ float linear_activate_kernel(float x){return x;}
__device__ float logistic_activate_kernel(float x){return 1.f/(1.f + expf(-x));}
__device__ float loggy_activate_kernel(float x){return 2.f/(1.f + expf(-x)) - 1;}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float elu_activate_kernel(float x){return (x >= 0)*x + (x < 0)*(expf(x)-1);}
__device__ float relie_activate_kernel(float x){return (x>0) ? x : .01f*x;}
__device__ float ramp_activate_kernel(float x){return x*(x>0)+.1f*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1f*x;}
__device__ float tanh_activate_kernel(float x){return (2.f/(1 + expf(-2*x)) - 1);}
__device__ float plse_activate_kernel(float x)
{
    if(x < -4) return .01f * (x + 4);
    if(x > 4)  return .01f * (x - 4) + 1;
    return .125f*x + .5f;
}
__device__ float stair_activate_kernel(float x)
{
    int n = floorf(x);
    if (n%2 == 0) return floorf(x/2);
    else return (x - n) + floorf(x/2);
}

__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_activate_kernel(x);
        case LOGISTIC:
            return logistic_activate_kernel(x);
        case LOGGY:
            return loggy_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case ELU:
            return elu_activate_kernel(x);
        case RELIE:
            return relie_activate_kernel(x);
        case RAMP:
            return ramp_activate_kernel(x);
        case LEAKY:
            return leaky_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
        case PLSE:
            return plse_activate_kernel(x);
        case STAIR:
            return stair_activate_kernel(x);
        case HARDTAN:
            return hardtan_activate_kernel(x);
        case LHTAN:
            return lhtan_activate_kernel(x);
    }
    return 0;
}

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = activate_kernel(x[i], a);
}

static void activate_array_gpu(float *x, int n, ACTIVATION a) 
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
    check_error(hipPeekAtLastError());
}

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
}

static void forward_maxpool_layer_gpu(float *l_output_gpu, float *input_gpu, int layer_out_w, int layer_out_h, int layer_batch, int layer_w, int layer_h, int layer_c, int layer_stride, int layer_size, int layer_pad)
{
    int h = layer_out_h;
    int w = layer_out_w;
    int c = layer_c;

    size_t n = h*w*c*layer_batch;

    forward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer_h, layer_w, layer_c, layer_stride, layer_size, layer_pad, input_gpu, l_output_gpu);
    check_error(hipPeekAtLastError());
}

static void forward_route_layer_25_gpu(float *l_output_gpu, float *input_l16)
{
    int offset = 0;
    float *input = input_l16;
    int input_size = 739328;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
}

static void forward_route_layer_28_gpu(float *l_output_gpu, float *input_l27, float *input_l24)
{
    int offset = 0;
    float *input = input_l27;
    int input_size = 92416;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
    offset += input_size;
    input = input_l24;
    input_size = 369664;
    copy_gpu(input_size, input, 1, l_output_gpu + offset, 1);
}

static void copy_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

static void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

static void forward_reorg_layer_gpu(float *l_output_gpu, float *input_gpu, int l_w, int l_h, int l_c, int l_batch, int l_stride)
{
    reorg_gpu(input_gpu, l_w, l_h, l_c, l_batch, l_stride, 0, l_output_gpu);
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
}

static void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int size = w*h*c*batch;
    reorg_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, w, h, c, batch, stride, forward, out);
    check_error(hipPeekAtLastError());
}

static void forward_region_layer_gpu(float *l_output_gpu, float *input_gpu, float *l_output, int l_batch, int l_inputs, int l_n, int l_w, int l_h, int l_coords, int l_background, int l_classes, int l_outputs)
{
    copy_gpu(l_batch*l_inputs, input_gpu, 1, l_output_gpu, 1);
    int b, n;
    for (b = 0; b < l_batch; ++b){
        for(n = 0; n < l_n; ++n){
            int index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, 0);
            activate_array_gpu(l_output_gpu + index, 2*l_w*l_h, LOGISTIC);
            index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, l_coords);
            activate_array_gpu(l_output_gpu + index,   l_w*l_h, LOGISTIC);
            index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, b, n*l_w*l_h, l_coords + 1);
        }
    }
    {
        int index = entry_index(l_w, l_h, l_outputs, l_coords, l_classes, 0, 0, l_coords + !l_background);
        softmax_gpu(input_gpu + index, l_classes + l_background, l_batch*l_n, l_inputs/l_n, l_w*l_h, 1, l_w*l_h, 1, l_output_gpu + index);
    }
    {
        cuda_pull_array(l_output_gpu, l_output, l_batch*l_outputs);
    }
}

static int entry_index(int l_w, int l_h, int l_outputs, int l_coords, int l_classes, int batch, int location, int entry)
{
    int n =   location / (l_w*l_h);
    int loc = location % (l_w*l_h);
    return batch*l_outputs + n*l_w*l_h*(l_coords+l_classes+1) + entry*l_w*l_h + loc;
}

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i*stride];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch*groups) return;
    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}

static void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    softmax_kernel<<<cuda_gridsize(batch*groups), BLOCK>>>(input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
    check_error(hipPeekAtLastError());
}

static void cuda_pull_array(float *x_gpu, float *x, size_t n)
{
    size_t size = sizeof(float)*n;
    hipError_t status = hipMemcpy(x, x_gpu, size, hipMemcpyDeviceToHost);
    check_error(status);
}

#if (1 == CHK_INTER_LAYER)
static void check_intermediate_layer_results(int l)
{
    int i, j, k;

    cudaMemcpy(spa_out_f32[l], sp_gpu_out_f32[l], sa_wid_s32[l] * sa_hei_s32[l] * sa_chn_s32[l] * sizeof(float), cudaMemcpyDeviceToHost);
    for(k = 0; k < sa_chn_s32[l]; k++)
    {
        for(j = 0; j < sa_hei_s32[l]; j++)
        {
            for(i = 0; i < sa_wid_s32[l]; i++)
            {
                if(fabsf(spa_out_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]] - spa_ref_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]]) > ACCEPTABLE_DIFF)
                {
                    printf("layer %d mismatch: w %d, h %d, c %d, out %f, GT %f\n", l, i, j, k, spa_out_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]], spa_ref_f32[l][i + j * sa_wid_s32[l] + k * sa_wid_s32[l] * sa_hei_s32[l]]);
                }
            }
        }
    }
}
#endif

