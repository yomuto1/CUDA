#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>

#include <time.h>
#include <math.h>
#include <stdio.h>

#define ITER_COUNT (5)

using namespace cv;
using namespace std;

static void img_resize_natC( unsigned char *p_out_u08, unsigned char *p_in_u08 );
static void gradient_magnitude_natC( unsigned char *p_out_u08, unsigned char *p_in_u08 );
static void img_resize_omp( unsigned char *p_out_u08, unsigned char *p_in_u08 );
static void gradient_magnitude_omp( unsigned char *p_out_u08, unsigned char *p_in_u08 );

__global__ void img_resize_cuda( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    if( (iCount_s32 < 800) && (jCount_s32 < 600) )
    {
        __shared__ unsigned char a_in_u08[32][8];

        a_in_u08[threadIdx_x_s32][threadIdx_y_s32] = p_in_u08[iCount_s32 + jCount_s32 * 800];

        __syncthreads();

        p_out_u08[iCount_s32 * 4 + jCount_s32 * 4 * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 1 + jCount_s32 * 4 * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 2 + jCount_s32 * 4 * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 3 + jCount_s32 * 4 * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + (jCount_s32 * 4 + 1) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 1 + (jCount_s32 * 4 + 1) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 2 + (jCount_s32 * 4 + 1) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 3 + (jCount_s32 * 4 + 1) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + (jCount_s32 * 4  + 2) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 1 + (jCount_s32 * 4 + 2) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 2 + (jCount_s32 * 4 + 2) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 3 + (jCount_s32 * 4 + 2) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + (jCount_s32 * 4 + 3) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 1 + (jCount_s32 * 4 + 3) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 2 + (jCount_s32 * 4 + 3) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
        p_out_u08[iCount_s32 * 4 + 3 + (jCount_s32 * 4 + 3) * 800 * 4] = a_in_u08[threadIdx_x_s32][threadIdx_y_s32];
    }
}

__global__ void gradient_magnitude_cuda( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    int threadIdx_x_s32 = threadIdx.x;
    int threadIdx_y_s32 = threadIdx.y;
    int iCount_s32 = blockIdx.x * blockDim.x + threadIdx_x_s32;
    int jCount_s32 = blockIdx.y * blockDim.y + threadIdx_y_s32;

    unsigned int gx_u32, gy_u32;    

    if( (iCount_s32 < 3200) && (jCount_s32 < 2400) )
    {
        __shared__ unsigned char a_in_u08[64 + 2][16 + 2];

        a_in_u08[threadIdx_x_s32 + 1][threadIdx_y_s32 + 1] = p_in_u08[iCount_s32 + jCount_s32 * 3200];
        if(0 == threadIdx_x_s32)
        {
            a_in_u08[threadIdx_x_s32][threadIdx_y_s32 + 1] = p_in_u08[iCount_s32 - 1 + jCount_s32 * 3200];
            if(0 == threadIdx_y_s32)
            {
                a_in_u08[threadIdx_x_s32][threadIdx_y_s32] = p_in_u08[iCount_s32 - 1 + (jCount_s32 - 1) * 3200];
            }
        }
        if(0 == threadIdx_y_s32)
        {
            a_in_u08[threadIdx_x_s32 + 1][threadIdx_y_s32] = p_in_u08[iCount_s32 + (jCount_s32 - 1) * 3200];
        }
        if(63 == threadIdx_x_s32)
        {
            a_in_u08[threadIdx_x_s32 + 2][threadIdx_y_s32 + 1] = p_in_u08[iCount_s32 + 1 + jCount_s32 * 3200];
            if(15 == threadIdx_y_s32)
            {
                a_in_u08[threadIdx_x_s32 + 2][threadIdx_y_s32 + 2] = p_in_u08[iCount_s32 + 1 + (jCount_s32 + 1) * 3200];
            }
        }
        if(15 == threadIdx_y_s32)
        {
            a_in_u08[threadIdx_x_s32 + 1][threadIdx_y_s32 + 2] = p_in_u08[iCount_s32 + (jCount_s32 + 1) * 3200];
        }

        __syncthreads();

        const unsigned char x_p1_u08 = a_in_u08[threadIdx_x_s32 + 2][threadIdx_y_s32 + 1];
        const unsigned char x_m1_u08 = a_in_u08[threadIdx_x_s32][threadIdx_y_s32 + 1];
        const unsigned char y_p1_u08 = a_in_u08[threadIdx_x_s32 + 1][threadIdx_y_s32 + 2];
        const unsigned char y_m1_u08 = a_in_u08[threadIdx_x_s32 + 1][threadIdx_y_s32];

        gx_u32 = abs(x_p1_u08 - x_m1_u08);
        gy_u32 = abs(y_p1_u08 - y_m1_u08);

        p_out_u08[iCount_s32 + jCount_s32 * 3200] = (unsigned char)(gx_u32 + gy_u32);
    }
}


int main(void)
{
    Mat image_in;
    static unsigned char sa_image_inter_u08[3200 * 2400];
    Mat image_out;
    static unsigned char sa_image_out_natC_u08[3200 * 2400];
    static unsigned char sa_image_out_CUDA_u08[3200 * 2400];

    clock_t time_init;
    clock_t time_sum = 0;

    unsigned int iCount_u32, jCount_u32;

    time_init = clock();
    time_init = clock() - time_init;

    cout <<  "Hello CUDA" << std::endl;

    image_in = imread( "../../../Pictures/PNG_transparency_demonstration_1.png", CV_LOAD_IMAGE_GRAYSCALE);

    if( image_in.data == NULL )
    {
        cout <<  "Could not open or find the image" << std::endl;
        
        return -1;
    }

    imwrite( "../../../Pictures/out_img_1.png", image_in );

    /* 4 times resize with CPU OpenCV */
    time_sum = clock();
    for( iCount_u32 = 0; iCount_u32 < ITER_COUNT; iCount_u32++ )
    {
        resize( image_in, image_out, Size(), 4.0, 4.0, CV_INTER_NN );
    }
    time_sum = clock() - time_init;

    cout <<  "LINEAR upscaling to 4x by CPU OpenCV " << (float)time_sum / CLOCKS_PER_SEC << std::endl;

    imwrite( "../../../Pictures/out_img_1_opencv_cpu.png", image_out );

    memset(sa_image_inter_u08, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));
    memset(image_out.data, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));

    /* 4 times resize with CPU natC */
    time_sum = clock();
    for( iCount_u32 = 0; iCount_u32 < ITER_COUNT; iCount_u32++ )
    {
        img_resize_natC( sa_image_inter_u08, image_in.data );
        gradient_magnitude_natC( image_out.data, sa_image_inter_u08 );
    }
    time_sum = clock() - time_init;

    cout <<  "LINEAR upscaling to 4x by CPU natC " << (float)time_sum / CLOCKS_PER_SEC << std::endl;

    memcpy(sa_image_out_natC_u08, image_out.data, 3200 * 2400 * sizeof(unsigned char));

    imwrite( "../../../Pictures/out_img_1_opencv_natC.png", image_out );

    memset(sa_image_inter_u08, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));
    memset(image_out.data, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));

    /* 4 times resize with CPU OMP */
    time_sum = clock();
    for( iCount_u32 = 0; iCount_u32 < ITER_COUNT; iCount_u32++ )
    {
        img_resize_omp( sa_image_inter_u08, image_in.data );
        gradient_magnitude_omp( image_out.data, sa_image_inter_u08 );
    }
    time_sum = clock() - time_init;

    cout <<  "LINEAR upscaling to 4x by GPU OMP " << (float)time_sum / CLOCKS_PER_SEC << std::endl;

    imwrite( "../../../Pictures/out_img_1_GPU_OMP.png", image_out );

    memset(sa_image_inter_u08, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));
    memset(image_out.data, 0, 800 * 600 * 4 * 4 * sizeof(unsigned char));

    /* 4 times resize with GPU CUDA */
    {
        unsigned char *p_image_in_u08;
        unsigned char *p_image_inter_u08;
        unsigned char *p_image_out_u08;

        dim3 grid_img_resize( 32, 8 );
        dim3 grid_numblocks_resize( 800 / grid_img_resize.x, 600 / grid_img_resize.y );
        dim3 grid_gradient_magnitude( 64, 16 );
        dim3 grid_numblocks_gradient( 3200 / grid_gradient_magnitude.x, 2400 / grid_gradient_magnitude.y);

        hipMalloc( (void**)&p_image_in_u08, 800 * 600 * sizeof(unsigned char) );
        hipMalloc( (void**)&p_image_inter_u08, 800 * 600 * 4 * 4 * sizeof(unsigned char) );
        hipMalloc( (void**)&p_image_out_u08, 800 * 600 * 4 * 4 * sizeof(unsigned char) );

        hipMemcpy( p_image_in_u08, image_in.data, 800 * 600 * sizeof(unsigned char), hipMemcpyHostToDevice );

        time_sum = clock();
        for( iCount_u32 = 0; iCount_u32 < ITER_COUNT; iCount_u32++ )
        {
            img_resize_cuda<<<grid_numblocks_resize, grid_img_resize>>>( p_image_inter_u08, p_image_in_u08 );
            gradient_magnitude_cuda<<<grid_numblocks_gradient, grid_gradient_magnitude>>>( p_image_out_u08, p_image_inter_u08 );
            hipMemcpy( image_out.data, p_image_out_u08, 800 * 600 * 4 * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost );
        }
        time_sum = clock() - time_init;

        hipMemcpy( image_out.data, p_image_out_u08, 800 * 600 * 4 * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost );

        cout <<  "LINEAR upscaling to 4x by GPU CUDA " << (float)time_sum / CLOCKS_PER_SEC << std::endl;

        memcpy(sa_image_out_CUDA_u08, image_out.data, 3200 * 2400 * sizeof(unsigned char));

        imwrite( "../../../Pictures/out_img_1_GPU_CUDA.png", image_out );

        for( jCount_u32 = 1; jCount_u32 < 2400 - 1; jCount_u32++ )
        {
            for( iCount_u32 = 1; iCount_u32 < 3200 - 1; iCount_u32++ )
            {
                if( sa_image_out_natC_u08[iCount_u32 + jCount_u32 * 3200] != sa_image_out_CUDA_u08[iCount_u32 + jCount_u32 * 3200] )
                {
                    printf("not matched: (%d, %d), %d, %d\n", iCount_u32, jCount_u32, sa_image_out_natC_u08[iCount_u32 + jCount_u32 * 3200], sa_image_out_CUDA_u08[iCount_u32 + jCount_u32 * 3200]);
                }
            }
        }

        hipFree( p_image_in_u08 );
        hipFree( p_image_inter_u08 );
        hipFree( p_image_out_u08 );
    }

    return 0;
}

static void img_resize_natC( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    unsigned int iCount_u32, jCount_u32;

    for( jCount_u32 = 0; jCount_u32 < 600; jCount_u32++ )
    {
        for( iCount_u32 = 0; iCount_u32 < 800; iCount_u32++ )
        {
            p_out_u08[iCount_u32 * 4 + 0 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
        }
    }
}

static void gradient_magnitude_natC( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    unsigned int iCount_u32, jCount_u32;
    unsigned char gx_u08, gy_u08;

    for( jCount_u32 = 1; jCount_u32 < 2400 - 1; jCount_u32++ )
    {
        for( iCount_u32 = 1; iCount_u32 < 3200 - 1; iCount_u32++ )
        {
            gx_u08 = abs(p_in_u08[iCount_u32 + 1 + jCount_u32 * 3200] - p_in_u08[iCount_u32 - 1 + jCount_u32 * 3200]);
            gy_u08 = abs(p_in_u08[iCount_u32 + (jCount_u32 + 1) * 3200] - p_in_u08[iCount_u32 + (jCount_u32 - 1) * 3200]);

            p_out_u08[iCount_u32 + jCount_u32 * 3200] = gx_u08 + gy_u08;
        }
    }
}

static void img_resize_omp( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    unsigned int iCount_u32, jCount_u32;

#pragma omp parallel for
    for( jCount_u32 = 0; jCount_u32 < 600; jCount_u32++ )
    {
#pragma omp parallel for
        for( iCount_u32 = 0; iCount_u32 < 800; iCount_u32++ )
        {
            p_out_u08[iCount_u32 * 4 + 0 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + jCount_u32 * 4 * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 1) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 2) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 0 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 1 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 2 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
            p_out_u08[iCount_u32 * 4 + 3 + (jCount_u32 * 4 + 3) * 800 * 4] = p_in_u08[iCount_u32 + jCount_u32 * 800];
        }
    }
}

static void gradient_magnitude_omp( unsigned char *p_out_u08, unsigned char *p_in_u08 )
{
    unsigned int iCount_u32, jCount_u32;
    unsigned char gx_u08, gy_u08;

#pragma omp parallel for
    for( jCount_u32 = 1; jCount_u32 < 2400 - 1; jCount_u32++ )
    {
#pragma omp parallel for
        for( iCount_u32 = 1; iCount_u32 < 3200 - 1; iCount_u32++ )
        {
            gx_u08 = abs(p_in_u08[iCount_u32 + 1 + jCount_u32 * 3200] - p_in_u08[iCount_u32 - 1 + jCount_u32 * 3200]);
            gy_u08 = abs(p_in_u08[iCount_u32 + (jCount_u32 + 1) * 3200] - p_in_u08[iCount_u32 + (jCount_u32 - 1) * 3200]);

            p_out_u08[iCount_u32 + jCount_u32 * 3200] = gx_u08 + gy_u08;
        }
    }
}

